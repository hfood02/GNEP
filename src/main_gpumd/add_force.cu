#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan and GPUMD development team
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
Add force to a group of atoms.
------------------------------------------------------------------------------*/

#include "add_force.cuh"
#include "model/atom.cuh"
#include "model/group.cuh"
#include "utilities/gpu_macro.cuh"
#include "utilities/read_file.cuh"
#include <iostream>
#include <vector>
#include <cstring>

static void __global__ add_force(
  const int group_size,
  const int group_size_sum,
  const int* g_group_contents,
  const double added_fx,
  const double added_fy,
  const double added_fz,
  double* g_fx,
  double* g_fy,
  double* g_fz)
{
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < group_size) {
    const int atom_id = g_group_contents[group_size_sum + tid];
    g_fx[atom_id] += added_fx;
    g_fy[atom_id] += added_fy;
    g_fz[atom_id] += added_fz;
  }
}

void Add_Force::compute(const int step, const std::vector<Group>& groups, Atom& atom)
{
  for (int call = 0; call < num_calls_; ++call) {
    const int step_mod_table_length = step % table_length_[call];
    const double added_fx = force_table_[call][0 * table_length_[call] + step_mod_table_length];
    const double added_fy = force_table_[call][1 * table_length_[call] + step_mod_table_length];
    const double added_fz = force_table_[call][2 * table_length_[call] + step_mod_table_length];
    const int num_atoms_total = atom.force_per_atom.size() / 3;
    const int group_size = groups[grouping_method_[call]].cpu_size[group_id_[call]];
    const int group_size_sum = groups[grouping_method_[call]].cpu_size_sum[group_id_[call]];
    add_force<<<(group_size - 1) / 64 + 1, 64>>>(
      group_size,
      group_size_sum,
      groups[grouping_method_[call]].contents.data(),
      added_fx,
      added_fy,
      added_fz,
      atom.force_per_atom.data(),
      atom.force_per_atom.data() + num_atoms_total,
      atom.force_per_atom.data() + num_atoms_total * 2);
    GPU_CHECK_KERNEL
  }
}

void Add_Force::parse(const char** param, int num_param, const std::vector<Group>& group)
{
  printf("Add force.\n");

  // check the number of parameters
  if (num_param != 6 && num_param != 4) {
    PRINT_INPUT_ERROR("add_force should have 5 or 3 parameters.\n");
  }

  // parse grouping method
  if (!is_valid_int(param[1], &grouping_method_[num_calls_])) {
    PRINT_INPUT_ERROR("grouping method should be an integer.\n");
  }
  if (grouping_method_[num_calls_] < 0) {
    PRINT_INPUT_ERROR("grouping method should >= 0.\n");
  }
  if (grouping_method_[num_calls_] >= group.size()) {
    PRINT_INPUT_ERROR("grouping method should < maximum number of grouping methods.\n");
  }

  // parse group id
  if (!is_valid_int(param[2], &group_id_[num_calls_])) {
    PRINT_INPUT_ERROR("group id should be an integer.\n");
  }
  if (group_id_[num_calls_] < 0) {
    PRINT_INPUT_ERROR("group id should >= 0.\n");
  }
  if (group_id_[num_calls_] >= group[grouping_method_[num_calls_]].number) {
    PRINT_INPUT_ERROR("group id should < maximum number of groups in the grouping method.\n");
  }

  printf(
    "    for atoms in group %d of grouping method %d.\n",
    group_id_[num_calls_],
    grouping_method_[num_calls_]);

  if (num_param == 6) {
    table_length_[num_calls_] = 1;
    force_table_[num_calls_].resize(table_length_[num_calls_] * 3);
    if (!is_valid_real(param[3], &force_table_[num_calls_][0])) {
      PRINT_INPUT_ERROR("fx should be a number.\n");
    }
    if (!is_valid_real(param[4], &force_table_[num_calls_][1])) {
      PRINT_INPUT_ERROR("fy should be a number.\n");
    }
    if (!is_valid_real(param[5], &force_table_[num_calls_][2])) {
      PRINT_INPUT_ERROR("fz should be a number.\n");
    }
    printf("    fx = %g eV/A.\n", force_table_[num_calls_][0]);
    printf("    fy = %g eV/A.\n", force_table_[num_calls_][1]);
    printf("    fz = %g eV/A.\n", force_table_[num_calls_][2]);
  } else {
    std::ifstream input(param[3]);
    if (!input.is_open()) {
      printf("Failed to open %s.\n", param[3]);
      exit(1);
    }

    std::vector<std::string> tokens = get_tokens(input);
    if (tokens.size() != 1) {
      PRINT_INPUT_ERROR("The first line of the add_force file should have 1 value.");
    }
    table_length_[num_calls_] = get_int_from_token(tokens[0], __FILE__, __LINE__);
    if (table_length_[num_calls_] < 2) {
      PRINT_INPUT_ERROR("Number of steps in the add_force file should >= 2.\n");
    } else {
      printf("    number of values in the add_force file = %d.\n", table_length_[num_calls_]);
    }

    force_table_[num_calls_].resize(table_length_[num_calls_] * 3);
    for (int n = 0; n < table_length_[num_calls_]; ++n) {
      std::vector<std::string> tokens = get_tokens(input);
      if (tokens.size() != 3) {
        PRINT_INPUT_ERROR("Number of force components at each step should be 3.");
      }
      for (int t = 0; t < 3; ++t) {
        force_table_[num_calls_][t * table_length_[num_calls_] + n] =
          get_double_from_token(tokens[t], __FILE__, __LINE__);
      }
    }
  }

  ++num_calls_;

  if (num_calls_ > 10) {
    PRINT_INPUT_ERROR("add_force cannot be used more than 10 times in one run.");
  }
}

void Add_Force::finalize() { num_calls_ = 0; }
