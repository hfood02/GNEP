#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan and GPUMD development team
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
The k-space part of the PPPM method.
------------------------------------------------------------------------------*/

#include "pppm.cuh"
#include "utilities/common.cuh"
#include "utilities/gpu_macro.cuh"
#include <cmath>
#include <vector>
#include <iostream>

namespace{

int get_best_K(const int m)
{
  int n = 16;
  while (n < m) {
    n *= 2;
  }
  return n;
}

__constant__ float sinc_coeff[6] = {1.0f, -1.6666667e-1f, 8.3333333e-3f, -1.9841270e-4f, 2.7557319e-6f, -2.5052108e-8f};

__device__ inline float sinc(const float x)
{
  float y = 0.0f;
  if (x * x <= 1.0f) {
    float term = 1.0f;
    for (int i = 0; i < 6; ++i) {
      y += sinc_coeff[i] * term;
      term *= x * x;
    }
  } else {
    y = sin(x) / x;
  }
  return y;
}

void __global__ find_k_and_G_opt(
  const PPPM::Para para,
  float* g_kx,
  float* g_ky,
  float* g_kz,
  float* g_G)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < para.K0K1K2) {
    int nk[3];
    nk[2] = n / para.K0K1;
    nk[1] = (n - nk[2] * para.K0K1) / para.K[0];
    nk[0] = n % para.K[0];

    // Eq. (6.40) in Allen & Tildesley
    float denominator[3] = {0.0f};
    for (int d = 0; d < 3; ++d) {
      if (nk[d] >= para.K_half[d]) {
        nk[d] -= para.K[d];
      }
      denominator[d] = sin(0.5f * para.two_pi_over_K[d] * nk[d]);
      denominator[d] *= denominator[d];
      denominator[d] = 1.0f - denominator[d] + 0.13333333f * denominator[d] * denominator[d];
      denominator[d] *= denominator[d];
    }
    const float kx = nk[0] * para.b[0][0] + nk[1] * para.b[1][0] + nk[2] * para.b[2][0];
    const float ky = nk[0] * para.b[0][1] + nk[1] * para.b[1][1] + nk[2] * para.b[2][1];
    const float kz = nk[0] * para.b[0][2] + nk[1] * para.b[1][2] + nk[2] * para.b[2][2];
    g_kx[n] = kx;
    g_ky[n] = ky;
    g_kz[n] = kz;
    const float ksq = kx * kx + ky * ky + kz * kz;

    // Eq. (6.39) in Allen & Tildesley
    float numerator = sinc(0.5f * para.two_pi_over_K[0] * nk[0]);
    numerator *= sinc(0.5f * para.two_pi_over_K[1] * nk[1]);
    numerator *= sinc(0.5f * para.two_pi_over_K[2] * nk[2]);
    numerator *= numerator * numerator;
    numerator *= numerator;

    // Eq. (6.41) in Allen & Tildesley
    if (ksq == 0.0f) {
      g_G[n] = 0.0f;
    } else {
      float G_opt = numerator * para.two_pi_over_V / ksq * exp(-ksq * para.alpha_factor);
      G_opt /= denominator[0] * denominator[1] * denominator[2];
      g_G[n] = G_opt;
    }
  }
}

void __global__ set_mesh_to_zero(const PPPM::Para para, hipfftComplex* g_mesh)
{
  const int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < para.K0K1K2) {
    g_mesh[n].x = 0.0f;
    g_mesh[n].y = 0.0f;
  }
}

__device__ inline int get_index_within_mesh(const int K, const int n)
{
  int y = n;
  if (n >= K) {
    y = n - K;
  } else if (n < 0) {
    y = n + K;
  }
  return y;
}

__global__ void find_mesh(
  const int N1,
  const int N2,
  const PPPM::Para para,
  const Box box,
  const float* g_charge,
  const double* g_x,
  const double* g_y,
  const double* g_z,
  hipfftComplex* g_mesh)
{
  const int n = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (n < N2) {
    const double x = g_x[n];
    const double y = g_y[n];
    const double z = g_z[n];
    const float q = g_charge[n];
    const float sx = (box.cpu_h[9] * x + box.cpu_h[10] * y + box.cpu_h[11] * z) * para.K[0];
    const float sy = (box.cpu_h[12] * x + box.cpu_h[13] * y + box.cpu_h[14] * z) * para.K[1];
    const float sz = (box.cpu_h[15] * x + box.cpu_h[16] * y + box.cpu_h[17] * z) * para.K[2];
    const int ix = int(sx + 0.5f); // can be 0, ..., K[0]
    const int iy = int(sy + 0.5f); // can be 0, ..., K[1]
    const int iz = int(sz + 0.5f); // can be 0, ..., K[2]
    const float dx = sx - ix; // (-0.5, 0.5)
    const float dy = sy - iy; // (-0.5, 0.5)
    const float dz = sz - iz; // (-0.5, 0.5)
    // Eq. (6.29) in Allen & Tildesley
    const float Wx[3] = {0.5f * (0.5f - dx) * (0.5f - dx), 0.75f - dx * dx, 0.5f * (0.5f + dx) * (0.5f + dx)};
    const float Wy[3] = {0.5f * (0.5f - dy) * (0.5f - dy), 0.75f - dy * dy, 0.5f * (0.5f + dy) * (0.5f + dy)};
    const float Wz[3] = {0.5f * (0.5f - dz) * (0.5f - dz), 0.75f - dz * dz, 0.5f * (0.5f + dz) * (0.5f + dz)};
    for (int n0 = -1; n0 <= 1; ++n0) {
      const int neighbor0 = get_index_within_mesh(para.K[0], ix + n0);  // can be 0, ..., K[0]-1
      for (int n1 = -1; n1 <= 1; ++n1) {
        const int neighbor1 = get_index_within_mesh(para.K[1], iy + n1);  // can be 0, ..., K[1]-1
        for (int n2 = -1; n2 <= 1; ++n2) {
          const int neighbor2 = get_index_within_mesh(para.K[2], iz + n2);  // can be 0, ..., K[2]-1
          const int neighbor012 = neighbor0 + para.K[0] * (neighbor1 + para.K[1] * neighbor2);
          const float W = Wx[n0 + 1] * Wy[n1 + 1] * Wz[n2 + 1];
          atomicAdd(&g_mesh[neighbor012].x, q * W);
        }
      }
    }
  }
}

void __global__ ik_times_mesh_times_G(
  const PPPM::Para para,
  const float* g_kx,
  const float* g_ky,
  const float* g_kz,
  const float* g_G,
  const hipfftComplex* g_mesh_fft,
  hipfftComplex* g_mesh_fft_x,
  hipfftComplex* g_mesh_fft_y,
  hipfftComplex* g_mesh_fft_z)
{
  const int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < para.K0K1K2) {
    const float kx = g_kx[n];
    const float ky = g_ky[n];
    const float kz = g_kz[n];
    const float G = g_G[n];
    hipfftComplex mesh_fft = g_mesh_fft[n];
    g_mesh_fft_x[n] = {mesh_fft.y * kx * G, -mesh_fft.x * kx * G};
    g_mesh_fft_y[n] = {mesh_fft.y * ky * G, -mesh_fft.x * ky * G};
    g_mesh_fft_z[n] = {mesh_fft.y * kz * G, -mesh_fft.x * kz * G};
  }
}

void __global__ find_mesh_G(
  const PPPM::Para para,
  const float* g_G,
  const hipfftComplex* g_mesh,
  hipfftComplex* g_mesh_G)
{
  const int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < para.K0K1K2) {
    const float G = g_G[n];
    hipfftComplex mesh = g_mesh[n];
    g_mesh_G[n] = {mesh.x * G, mesh.y * G};
  }
}

__global__ void find_force_from_field(
  const int N1,
  const int N2,
  const PPPM::Para para,
  const Box box,
  const float* g_charge,
  const double* g_x,
  const double* g_y,
  const double* g_z,
  const hipfftComplex* g_mesh_G,
  const hipfftComplex* g_mesh_fft_x_ifft,
  const hipfftComplex* g_mesh_fft_y_ifft,
  const hipfftComplex* g_mesh_fft_z_ifft,
  float* g_D_real,
  double* g_fx,
  double* g_fy,
  double* g_fz)
{
  const int n = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (n < N2) {
    const double x = g_x[n];
    const double y = g_y[n];
    const double z = g_z[n];
    const float q = K_C_SP * g_charge[n] * 2.0f;
    const float sx = (box.cpu_h[9] * x + box.cpu_h[10] * y + box.cpu_h[11] * z) * para.K[0];
    const float sy = (box.cpu_h[12] * x + box.cpu_h[13] * y + box.cpu_h[14] * z) * para.K[1];
    const float sz = (box.cpu_h[15] * x + box.cpu_h[16] * y + box.cpu_h[17] * z) * para.K[2];
    const int ix = int(sx + 0.5f); // can be 0, ..., K[0]
    const int iy = int(sy + 0.5f); // can be 0, ..., K[1]
    const int iz = int(sz + 0.5f); // can be 0, ..., K[2]
    const float dx = sx - ix; // (-0.5, 0.5)
    const float dy = sy - iy; // (-0.5, 0.5)
    const float dz = sz - iz; // (-0.5, 0.5)
    // Eq. (6.29) in Allen & Tildesley
    const float Wx[3] = {0.5f * (0.5f - dx) * (0.5f - dx), 0.75f - dx * dx, 0.5f * (0.5f + dx) * (0.5f + dx)};
    const float Wy[3] = {0.5f * (0.5f - dy) * (0.5f - dy), 0.75f - dy * dy, 0.5f * (0.5f + dy) * (0.5f + dy)};
    const float Wz[3] = {0.5f * (0.5f - dz) * (0.5f - dz), 0.75f - dz * dz, 0.5f * (0.5f + dz) * (0.5f + dz)};
    float D_real = 0.0f;
    float E[3] = {0.0f, 0.0f, 0.0f};
    for (int n0 = -1; n0 <= 1; ++n0) {
      const int neighbor0 = get_index_within_mesh(para.K[0], ix + n0);  // can be 0, ..., K[0]-1
      for (int n1 = -1; n1 <= 1; ++n1) {
        const int neighbor1 = get_index_within_mesh(para.K[1], iy + n1);  // can be 0, ..., K[1]-1
        for (int n2 = -1; n2 <= 1; ++n2) {
          const int neighbor2 = get_index_within_mesh(para.K[2], iz + n2);  // can be 0, ..., K[2]-1
          const int neighbor012 = neighbor0 + para.K[0] * (neighbor1 + para.K[1] * neighbor2);
          const float W = Wx[n0 + 1] * Wy[n1 + 1] * Wz[n2 + 1];
          D_real += W * g_mesh_G[neighbor012].x;
          E[0] += W * g_mesh_fft_x_ifft[neighbor012].x;
          E[1] += W * g_mesh_fft_y_ifft[neighbor012].x;
          E[2] += W * g_mesh_fft_z_ifft[neighbor012].x;
        }
      }
    }
    g_D_real[n] = 2.0f * K_C_SP * D_real;
    g_fx[n] += q * E[0];
    g_fy[n] += q * E[1];
    g_fz[n] += q * E[2];
  } 
}

void __global__ find_potential_and_virial(
  const int N,
  const PPPM::Para para,
  const hipfftComplex* g_S,
  const float* g_kx,
  const float* g_ky,
  const float* g_kz,
  const float* g_G,
  double* g_virial,
  double* g_pe)
{
  const int tid = threadIdx.x;
  int number_of_batches = (para.K0K1K2 - 1) / 1024 + 1;
  __shared__ float s_data[1024];
  float data = 0.0f;

  for (int batch = 0; batch < number_of_batches; ++batch) {
    const int n = tid + batch * 1024;
    if (n < para.K0K1K2) {
      hipfftComplex S = g_S[n];
      const float GSS = g_G[n] * (S.x * S.x + S.y * S.y);
      const float kx = g_kx[n];
      const float ky = g_ky[n];
      const float kz = g_kz[n];
      const float ksq = kx * kx + ky * ky + kz * kz;
      if (ksq != 0.0f) {
        const float alpha_k_factor = 2.0f * para.alpha_factor + 2.0f / ksq;
        switch (blockIdx.x) {
          case 0:
          data += GSS * (1.0f - alpha_k_factor * kx * kx); // xx
          break;
          case 1:
          data += GSS * (1.0f - alpha_k_factor * ky * ky); // yy
          break;
          case 2:
          data += GSS * (1.0f - alpha_k_factor * kz * kz); // zz
          break;
          case 3:
          data -= GSS * (alpha_k_factor * kx * ky); // xy
          break;
          case 4:
          data -= GSS * (alpha_k_factor * ky * kz); // yz
          break;
          case 5:
          data -= GSS * (alpha_k_factor * kz * kx); // zx
          break;
          case 6:
          data += GSS; // potential
          break;
        }
      }
    }
  }
  s_data[tid] = data;
  __syncthreads();

  for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
    if (tid < offset) {
      s_data[tid] += s_data[tid + offset];
    }
    __syncthreads();
  }

  number_of_batches = (N - 1) / 1024 + 1;
  for (int batch = 0; batch < number_of_batches; ++batch) {
    const int n = tid + batch * 1024;
    if (n < N) {
      // virial order
      // xx xy xz    0 3 4
      // yx yy yz    6 1 5
      // zx zy zz    7 8 2
      switch (blockIdx.x) {
        case 0:
          g_virial[n + 0 * N] += s_data[0] * para.potential_factor; // xx
          break;
        case 1:
          g_virial[n + 1 * N] += s_data[0] * para.potential_factor; // yy
          break;
        case 2:
          g_virial[n + 2 * N] += s_data[0] * para.potential_factor; // zz
          break;
        case 3:
          g_virial[n + 3 * N] += s_data[0] * para.potential_factor; // xy
          g_virial[n + 6 * N] += s_data[0] * para.potential_factor; // yx
          break;
        case 4:
          g_virial[n + 5 * N] += s_data[0] * para.potential_factor; // yz
          g_virial[n + 8 * N] += s_data[0] * para.potential_factor; // zy
          break;
        case 5:
          g_virial[n + 4 * N] += s_data[0] * para.potential_factor; // xz
          g_virial[n + 7 * N] += s_data[0] * para.potential_factor; // zx
          break;
        case 6:
          g_pe[n] += s_data[0] * para.potential_factor;
          break;
      }
    }
  }
}

}

PPPM::PPPM()
{
  // nothing
}

PPPM::~PPPM()
{
  hipfftDestroy(plan);
}

void PPPM::allocate_memory()
{
  kx.resize(para.K0K1K2);
  ky.resize(para.K0K1K2);
  kz.resize(para.K0K1K2);
  G.resize(para.K0K1K2);
  mesh.resize(para.K0K1K2);
  mesh_G.resize(para.K0K1K2);
  mesh_x.resize(para.K0K1K2);
  mesh_y.resize(para.K0K1K2);
  mesh_z.resize(para.K0K1K2);
  // para.K[2] is the slowest changing dimension; para.K[0] is the fastest changing dimension
  if (hipfftPlan3d(&plan, para.K[2], para.K[1], para.K[0], HIPFFT_C2C) != HIPFFT_SUCCESS) {
    std::cout << "CUFFT error: Plan creation failed" << std::endl;
    exit(1);
  }
}

void PPPM::initialize(const float alpha_input)
{
  para.alpha = alpha_input;
  para.alpha_factor = 0.25f / (para.alpha * para.alpha);
  para.K[0] = 16;
  para.K[1] = 16;
  para.K[2] = 16;
  para.K0K1K2 = para.K[0] * para.K[1] * para.K[2];
  allocate_memory();
}

void PPPM::find_para(const int N, const Box& box)
{
  const float two_pi = 6.2831853f;
  const double mesh_spacing = 1.0; // Is this good enough?
  const double volume = box.get_volume();
  para.two_pi_over_V = two_pi / volume;
  int K[3] = {0};
  for (int d = 0; d < 3; ++d) {
    const double box_thickness = volume / box.get_area(d);
    K[d] = box_thickness / mesh_spacing;
    K[d] = get_best_K(K[d]);
    para.K_half[d] = K[d] / 2;
    para.two_pi_over_K[d] = two_pi / K[d];
  }
  para.K0K1 = K[0] * K[1];
  para.K0K1K2 = para.K0K1 * K[2];
  if (K[0] != para.K[0] || K[1] != para.K[1] || K[2] != para.K[2]) {
    para.K[0] = K[0];
    para.K[1] = K[1];
    para.K[2] = K[2];
    allocate_memory();
  }
  para.potential_factor = K_C_SP / N;
  for (int d = 0; d < 3; ++d) {
    para.b[0][d] = two_pi * (float)box.cpu_h[9 + d];
    para.b[1][d] = two_pi * (float)box.cpu_h[12 + d];
    para.b[2][d] = two_pi * (float)box.cpu_h[15 + d];
  }
}

void PPPM::find_force(
  const int N,
  const int N1,
  const int N2,
  const Box& box,
  const GPU_Vector<float>& charge,
  const GPU_Vector<double>& position_per_atom,
  GPU_Vector<float>& D_real,
  GPU_Vector<double>& force_per_atom,
  GPU_Vector<double>& virial_per_atom,
  GPU_Vector<double>& potential_per_atom)
{
  find_para(N, box);

  find_k_and_G_opt<<<(para.K0K1K2 - 1) / 64 + 1, 64>>>(
    para, 
    kx.data(), 
    ky.data(), 
    kz.data(), 
    G.data());
  GPU_CHECK_KERNEL

  set_mesh_to_zero<<<(para.K0K1K2 - 1) / 64 + 1, 64>>>(para, mesh.data());
  GPU_CHECK_KERNEL

  find_mesh<<<(N - 1) / 64 + 1, 64>>>(
    N1,
    N2,
    para,
    box,
    charge.data(),
    position_per_atom.data(),
    position_per_atom.data() + N,
    position_per_atom.data() + N * 2,
    mesh.data());
  GPU_CHECK_KERNEL

  if (hipfftExecC2C(plan, mesh.data(), mesh.data(), HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
    std::cout << "CUFFT error: ExecC2C Forward failed" << std::endl;
    exit(1);
  }

  ik_times_mesh_times_G<<<(para.K0K1K2 - 1) / 64 + 1, 64>>>(
    para,
    kx.data(),
    ky.data(),
    kz.data(),
    G.data(),
    mesh.data(),
    mesh_x.data(),
    mesh_y.data(),
    mesh_z.data());
  GPU_CHECK_KERNEL

  find_mesh_G<<<(para.K0K1K2 - 1) / 64 + 1, 64>>>(
    para,
    G.data(),
    mesh.data(),
    mesh_G.data());
  GPU_CHECK_KERNEL


  if (hipfftExecC2C(plan, mesh_G.data(), mesh_G.data(), HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
    std::cout << "CUFFT error: ExecC2C Inverse failed" << std::endl;
    exit(1);
  }

  if (hipfftExecC2C(plan, mesh_x.data(), mesh_x.data(), HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
    std::cout << "CUFFT error: ExecC2C Inverse failed" << std::endl;
    exit(1);
  }

  if (hipfftExecC2C(plan, mesh_y.data(), mesh_y.data(), HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
    std::cout << "CUFFT error: ExecC2C Inverse failed" << std::endl;
    exit(1);
  }

  if (hipfftExecC2C(plan, mesh_z.data(), mesh_z.data(), HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
    std::cout << "CUFFT error: ExecC2C Inverse failed" << std::endl;
    exit(1);
  }

  find_force_from_field<<<(N - 1) / 64 + 1, 64>>>(
    N1,
    N2,
    para,
    box,
    charge.data(),
    position_per_atom.data(),
    position_per_atom.data() + N,
    position_per_atom.data() + N * 2,
    mesh_G.data(),
    mesh_x.data(),
    mesh_y.data(),
    mesh_z.data(),
    D_real.data(),
    force_per_atom.data(),
    force_per_atom.data() + N,
    force_per_atom.data() + N * 2);
  GPU_CHECK_KERNEL

  find_potential_and_virial<<<7, 1024>>>(
    N,
    para,
    mesh.data(),
    kx.data(),
    ky.data(),
    kz.data(),
    G.data(),
    virial_per_atom.data(),
    potential_per_atom.data());
  GPU_CHECK_KERNEL
}
